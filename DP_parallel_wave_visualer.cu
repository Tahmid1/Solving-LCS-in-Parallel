#include "hip/hip_runtime.h"
/* DP_parallel_wave_visualer.cu
 * implemented for CS87 final project
 * by Brandon Chow and Tahmid Rahman
 * implements an animated LCS solver on the GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include "myopengllib.h"
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <math.h>
#include <sys/types.h>
#include <signal.h>

static int N       = 256;

typedef struct my_cuda_data {
  int* read_grid;
  hiprandState *dev_random;
  int N;
  int num_iters;
  int cur_iters;
  hipEvent_t start;
  hipEvent_t stop;
  int* string_a;
  int* string_b;
  int* currMax;
} my_cuda_data;
my_cuda_data cudaData;

static void clean_up(void);
static void compute_LCS(uchar4 *devPtr, void *my_data) ;
__global__ void  LCS_kernel(my_cuda_data data);
__global__ void  retrace_kernel(my_cuda_data data);
__global__ void color_in_kernel(uchar4 *optr, my_cuda_data data);
__device__ int getFireNeighbor(int* data, int x, int y, int N);
__device__ void colorPix(uchar4 *optr, int offset, int type);

void usage();

/**************************************************************/
//MAIN
/**************************************************************/
int main(int argc, char *argv[])  {
  int i, j;
  //process_args(argc, argv);
  //int world[N][N];

  //parse input
  if (argc != 1){
    if(argc != 2) {
      printf("usage: ./dp_rand (size) \n");
      exit(0);
    } else {
      N = atoi(argv[1]);
    }
  }
  N = N+1;
  int* world;
  int* string_a = new int[N-1];
  int* string_b = new int[N-1];


  for (int i = 0; i < N-1; i++) {
    string_a[i] = rand() % 26;
  }


  for (int i = 0; i < N-1; i++) {
    string_b[i] = rand()%26;
  }

  //set up world
  world = new int[N*N]();

  hipEventCreate(&(cudaData.start));
  hipEventCreate(&(cudaData.stop));

  GPUDisplayData my_display(N, N, &cudaData, "Simple openGL-Cuda");

  //make world
  //fill in trees
  for(i=0; i < N; i++){
    for(j=0; j < N; j++){
      world[i*N + j] = -1*i + -1*j;
    }
  }

  cudaData.num_iters = 2*N;
  cudaData.cur_iters = 0;
  cudaData.N = N;

  int x = -1;
  int* currMax;
  currMax = &x;

  /*
     printf("%c\n", cudaData.string_a[40]);
     printf("%s\n", cudaData.string_b);
   */

  //allocate memory for currMax
  HANDLE_ERROR(hipMalloc((void**)&cudaData.currMax,
        sizeof(int)), "malloc read_grid") ;

  // copy the initial data to the GPU
  HANDLE_ERROR (hipMemcpy(cudaData.currMax, currMax,
        sizeof(int), hipMemcpyHostToDevice), "copy dev_grid to GPU") ; 

  //allocate memory for grid
  HANDLE_ERROR(hipMalloc((void**)&cudaData.read_grid,
        sizeof(int)*N*N), "malloc read_grid") ; 

  // copy the initial data to the GPU
  HANDLE_ERROR (hipMemcpy(cudaData.read_grid, world,
        sizeof(int)*N*N, hipMemcpyHostToDevice), "copy read_grid to GPU") ;

  //allocate memory for string_a
  HANDLE_ERROR(hipMalloc((void**)&cudaData.string_a,
        sizeof(int)*(N-1)), "malloc read_grid") ; 

  // copy the initial data to the GPU
  HANDLE_ERROR (hipMemcpy(cudaData.string_a, string_a,
        sizeof(int)*(N-1), hipMemcpyHostToDevice), "copy string_a to GPU") ;


  //allocate memory for string_b
  HANDLE_ERROR(hipMalloc((void**)&cudaData.string_b,
        sizeof(int)*(N-1)), "malloc read_grid")

    // copy the initial data to the GPU
    HANDLE_ERROR (hipMemcpy(cudaData.string_b, string_b,
          sizeof(int)*(N-1), hipMemcpyHostToDevice), "copy string_b to GPU") ; 


  // register a clean-up function on exit that will call hipFree
  // on any hipMalloc'ed space
  my_display.RegisterExitFunction(clean_up);

  // have the library run our Cuda animation
  my_display.AnimateComputation(compute_LCS);

  return 0;
}//end main

/**************************************************************/
//FUNCTION: clean_up
// passed to AnimateComputation method.
// it is called when the program exits and should clean up
// all hipMalloc'ed state.  Your clean-up function's prototype
// must match this, which means you need a global(s) that point
// to anything hipMalloc'ed so you can free them here
/**************************************************************/
static void clean_up(void) {
  hipFree(cudaData.read_grid);
  hipFree(cudaData.dev_random);
}


/* computeLCS
 * @ inputs: devPtr
 * @ inputs: my_data - a struct consisting of important data for LCS
 * 
 * This function computes the LCS of two strings.
 */
static void compute_LCS(uchar4 *devPtr, void *my_data) {
  //printf("INSIDE ANIMATE FIRE\n");
  char place_holder;

  //divvy up blocks and threads
  my_cuda_data * cudaData = (my_cuda_data *)my_data;

  //count iterations
  int c_iters = cudaData->cur_iters;
  cudaData->cur_iters = c_iters + 1;

  int N = cudaData->N;

  int thread_count;
  //printf("HERE HERE HERE\n");
  //char* A = cudaData->string_a;
  //printf("%c\n", A[2]);
  //printf("HERE HERE HERE\n");
  thread_count = 32;

  //set up blocks
  dim3 blocks(N/thread_count,N/thread_count,1);
  dim3 threads_block(thread_count,thread_count,1);

  //if we've done the max number of iterations, handle timing info and quit
  if (c_iters < 2* N + 1){
    LCS_kernel<<<blocks, threads_block>>>(*cudaData);
    color_in_kernel<<<blocks,threads_block>>>(devPtr, *cudaData);
  }
  else if (c_iters < 4 * N + 2){
    retrace_kernel<<<blocks, threads_block>>>(*cudaData);
    color_in_kernel<<<blocks,threads_block>>>(devPtr, *cudaData);
  }
  else{
    //find the colors of cells
    float tim;
    hipEventRecord(cudaData->stop, 0);
    hipEventSynchronize(cudaData->stop);
    hipEventElapsedTime(&tim, cudaData->start, cudaData->stop);
    //clean_up();

    scanf("%c", &place_holder);

    kill(getpid(), SIGKILL);
  }
}//end animate_fire


/* color_in_kernel
 * @ inputs: optr
 * @ inputs: data - cuda data struct for holding useful info
 * 
 * kernel dedicated to coloring cells in
 */
__global__ void color_in_kernel(uchar4 *optr, my_cuda_data data) {
  //get the offset so we can map from 2d array into 1d
  const int* fire_state = data.read_grid;
  int N = data.N;

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int offset = x + y*N;

  //change pixel colors based off world values
  int state;
  state = fire_state[offset];

  if(state < 0){ //cell not yet filled in
    colorPix(optr, offset, 0);
  }

  else if(state > N){//cell filled in 
    colorPix(optr, offset, 2);
  }

  else{
    colorPix(optr, offset, 1);
  }
}// end fire_to_color


/* LCS_kernel
 * @ inputs: data - a struct containing useful data for solving LCS 
 * 
 * this function either:
 *    increments a cell's age by 1
 *    computes the LCS value of a cell based on the LCS algorithm if age = 0
 */
__global__ void  LCS_kernel(my_cuda_data data){

  int N = data.N;
  int* read_data = data.read_grid;
  int* string_a = data.string_a;
  int* string_b = data.string_b;

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int offset = x + y*N;

  int state = read_data[offset];

  if (state < 0){
    read_data[offset] = state+1;
  }

  else if (state > 0){
  }

  else{
    if (x == 0){
      //printf("basecasing x\n");
      read_data[offset] = 0;
    }
    else if (y == 0){
      //printf("basecasing y\n");
      read_data[offset] = 0;
    }

    else if (string_a[x] == string_b[y]){
      //printf("Found matching letters\n");
      int old_offset = (x-1) + (y-1)*N; 
      read_data[offset] = read_data[old_offset] + 1;
    }

    else{
      //printf("maximizing from old\n");
      int old_offset1 = (x-1) + y*N;
      int old_offset2 = x + (y-1)*N;
      int max_sublen1 = read_data[old_offset1];
      int max_sublen2 = read_data[old_offset2];

      if (max_sublen1 > max_sublen2){
        read_data[offset] = max_sublen1;
      }
      else{
        read_data[offset] = max_sublen2;
      }
    }
  }
}//end fire_sim_kernel


/* retrace_kernel
 * @ inputs: data - a struct containing useful data for solving LCS 
 * 
 * retraces the final LCS value backwards
 */
__global__ void  retrace_kernel(my_cuda_data data){
  int N = data.N;
  int* read_data = data.read_grid;
  int* string_a = data.string_a;
  int* string_b = data.string_b;

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int offset = x + y*N;

  int state = read_data[offset];

  if ((x == (N-2)) && (y == (N-2))){
    read_data[offset] += N + 1;
  }
  if (state > N){
    if (string_a[x] == string_b[y]){
      int new_offset = (x-1) + (y-1)*N;
      read_data[new_offset] += N + 1;
    }
    else{
      int old_offset1 = (x-1) + y*N;
      int old_offset2 = x + (y-1)*N;
      int max_sublen1 = read_data[old_offset1];
      int max_sublen2 = read_data[old_offset2];

      if (max_sublen1 > max_sublen2){
        read_data[old_offset1] += N + 1;
      }
      else{
        read_data[old_offset2] += N + 1; 
      }     
    }
  }
} 

/*colorPix
 * @ inputs: optr
 * @ inputs: offset - offset to denote cell being colored
 * @ inputs: type - denotes the color to be set based on type
 * 
 *colors a pixel
 *type determines if it is an unfilled cell (0), filled cell(1),
 */
__device__ void colorPix(uchar4 *optr, int offset, int type){
  int x = 0, y = 0, z = 0;
  if (type == 0){
    z = 255;
  }
  else if (type == 1){
    y = 255;
  }
  else if (type == 2){
    x = 255;
  }

  optr[offset].x = x;   //R
  optr[offset].y = y;   //G
  optr[offset].z = z; //B
  optr[offset].w = 255; //opacity
}
