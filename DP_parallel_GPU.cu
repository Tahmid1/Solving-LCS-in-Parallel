#include "hip/hip_runtime.h"
/* DP_parallel_GPU.cu
 * by Brandon Chow and Tahmid Rahman
 * implemented for CS87 final project
 * uses GPU to solve LCS using parallel Waveform
 */

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include "myopengllib.h"
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <math.h>
#include <sys/types.h>
#include <signal.h>

static int N       = 512;

//struct for storing useful information to pass into device
typedef struct my_cuda_data {
  int* read_grid;
  hiprandState *dev_random;
  int N;
  int num_iters;
  int cur_iters;
  hipEvent_t start;
  hipEvent_t stop;
  int* string_a;
  int* string_b;
  int currMax;

} my_cuda_data;
my_cuda_data cudaData;

static void clean_up(void);
static void compute_LCS(uchar4 *devPtr, void *my_data);
__global__ void  LCS_kernel(my_cuda_data data);
void usage();



//main function
int main(int argc, char *argv[])  {
  int i, j;
  //process_args(argc, argv);
  //int world[N][N];

  //parse input
  if (argc != 1){
    if(argc != 2) {
      printf("usage: ./dp_rand (size) \n");
      exit(0);
    } else {
      N = atoi(argv[1]);
    }
  }
  N = N+1;
  int* world;
  int* string_a = new int[N-1];
  int* string_b = new int[N-1];


  for (int i = 0; i < N-1; i++) {
    string_a[i] = rand() % 26;
  }


  for (int i = 0; i < N-1; i++) {
    string_b[i] = rand()%26;
  }

  //set up world
  world = new int[N*N]();

  hipEventCreate(&(cudaData.start));
  hipEventCreate(&(cudaData.stop));

  GPUDisplayData my_display(N, N, &cudaData, "Simple openGL-Cuda");

  //preset table to start off with negative numbers associated with
  //when the cell should "wake up"
  for(i=0; i < N; i++){
    for(j=0; j < N; j++){
      world[i*N + j] = -1*i + -1*j;
    }
  }

  cudaData.num_iters = 2*N;
  cudaData.cur_iters = 0;
  cudaData.N = N;
  cudaData.currMax = 0; 

  //allocate memory for grid
  HANDLE_ERROR(hipMalloc((void**)&cudaData.read_grid,
        sizeof(int)*N*N), "malloc read_grid") ; 

  // copy the initial data to the GPU
  HANDLE_ERROR (hipMemcpy(cudaData.read_grid, world,
        sizeof(int)*N*N, hipMemcpyHostToDevice), "copy read_grid to GPU") ;

  //allocate memory for string_a
  HANDLE_ERROR(hipMalloc((void**)&cudaData.string_a,
        sizeof(int)*(N-1)), "malloc read_grid") ; 

  // copy the initial data to the GPU
  HANDLE_ERROR (hipMemcpy(cudaData.string_a, string_a,
        sizeof(int)*(N-1), hipMemcpyHostToDevice), "copy string_a to GPU") ;


  //allocate memory for string_b
  HANDLE_ERROR(hipMalloc((void**)&cudaData.string_b,
        sizeof(int)*(N-1)), "malloc read_grid")

    // copy the initial data to the GPU
    HANDLE_ERROR (hipMemcpy(cudaData.string_b, string_b,
          sizeof(int)*(N-1), hipMemcpyHostToDevice), "copy string_b to GPU") ; 


  // register a clean-up function on exit that will call hipFree
  // on any hipMalloc'ed space
  my_display.RegisterExitFunction(clean_up);

  // have the library run our Cuda animation
  my_display.AnimateComputation(compute_LCS);

  return 0;
}//end main


/* clean_up
 *
 * passed to RegisterExitFunction
 * it is called when the program exits and should clean up
 * all hipMalloc'ed state.
 */ 
static void clean_up(void) {
  hipFree(cudaData.read_grid);
  hipFree(cudaData.dev_random);
}


/* computeLCS
 * @ inputs: devPtr
 * @ inputs: my_data - a struct consisting of important data for LCS
 * 
 * This function computes the LCS of two strings.
 */
static void compute_LCS(uchar4 *devPtr, void *my_data) {
  //printf("INSIDE ANIMATE FIRE\n");
  char place_holder;

  //divvy up blocks and threads
  my_cuda_data * cudaData = (my_cuda_data *)my_data;

  //count iterations
  int c_iters = cudaData->cur_iters;
  cudaData->cur_iters = c_iters + 1;

  int thread_count;

  //if we've done the max number of iterations, handle timing info and quit
  if (c_iters == cudaData->num_iters){
    float tim;
    hipEventRecord(cudaData->stop, 0);
    hipEventSynchronize(cudaData->stop);
    hipEventElapsedTime(&tim, cudaData->start, cudaData->stop);
    //clean_up();
    printf ("\n\nThe animation took: %f s\n\n", tim/1000);

    printf("\nPress any key to exit.\n");

    scanf("%c", &place_holder);

    kill(getpid(), SIGKILL);
  }


  thread_count = 512;

  //set up blocks
  dim3 blocks(N/thread_count,N/thread_count,1);
  dim3 threads_block(thread_count,thread_count,1);

  int N = cudaData->N;
  float tim;
  hipEventRecord(cudaData->start, 0);

  //update cells
  for (int i = 0; i < 2*N; i++){
    LCS_kernel<<<blocks, threads_block>>>(*cudaData);
  }

  hipEventRecord(cudaData->stop, 0);
  hipEventSynchronize(cudaData->stop);
  hipEventElapsedTime(&tim, cudaData->start, cudaData->stop);
  //clean_up();
  printf ("\n\nThe animation took: %f s\n\n", tim/1000);

  printf("\nPress any key to exit.\n");

  scanf("%c", &place_holder);

  kill(getpid(), SIGKILL);

}

/* LCS_kernel
 * @ inputs: data - a struct containing useful data for solving LCS 
 * 
 * this function either:
 *    increments a cell's age by 1
 *    computes the LCS value of a cell based on the LCS algorithm if age = 0
 */
__global__ void  LCS_kernel(my_cuda_data data){

  int N = data.N;
  int* read_data = data.read_grid;
  int* string_a = data.string_a;
  int* string_b = data.string_b;

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int offset = x + y*N;

  int state = read_data[offset];

  if (state < 0){
    read_data[offset] = state+1;
  }

  else if (state == 0){
    if (x == 0){
      read_data[offset] = 0;
    }
    else if (y == 0){
      read_data[offset] = 0;
    }

    else if (string_a[x] == string_b[y]){
      int old_offset = (x-1) + (y-1)*N; 
      read_data[offset] = read_data[old_offset] + 1;
    }

    else{
      int old_offset1 = (x-1) + y*N;
      int old_offset2 = x + (y-1)*N;
      int max_sublen1 = read_data[old_offset1];
      int max_sublen2 = read_data[old_offset2];

      if (max_sublen1 > max_sublen2){
        read_data[offset] = max_sublen1;
      }
      else{
        read_data[offset] = max_sublen2;
      }
    }
  }
}
